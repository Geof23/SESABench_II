#include "hip/hip_runtime.h"
#include "stdio.h"

//must be at least 2
#define NUM 4
// #define NUM 64 

__shared__ int shared[NUM];

__device__ inline void swap(int & a, int & b) {
  // Alternative swap doesn't use a temporary register:
  // a ^= b;
  // b ^= a;
  // a ^= b;
  
  int tmp = a;
  a = b;
  b = tmp;
}

__global__ void BitonicKernel(int * values)
{
  unsigned int tid = threadIdx.x;
  // Copy input to shared mem.  
  shared[tid] = values[tid];
  printf("tid: %d, blockDim: %d\n", tid, blockDim.x);
  __syncthreads(); //syncthreads 1

  // Parallel bitonic sort.
  for (unsigned int k = 2; k <= blockDim.x; k *= 2) {
    for (unsigned int j = k / 2; j>0; j /= 2) {
      unsigned int ixj = tid ^ j;
      if (ixj > tid) { //if1
	if ((tid & k) == 0) { //if2
	  if (shared[tid] > shared[ixj])  //if3
	    swap(shared[tid], shared[ixj]);
	}
	else {
	  if (shared[tid] < shared[ixj]) //if4
	    swap(shared[tid], shared[ixj]);
	}
      }
      __syncthreads(); //syncthreads2
    }
  } //end sort
  
  // Write result.
  values[tid] = shared[tid];
}


int main() {
  //int *values = (int *)malloc(sizeof(int) * NUM); 
  int values[NUM];
  // the following is equivalent to calling the kernel using <<<...>>>(BitonicKernel)
   klee_make_symbolic(values, sizeof(int)*NUM, "values");
  // for(int x = 2; x < NUM; ++x) values[x] = NUM - x;
  // values[0] = klee_int( "values0" );
  // values[1] = klee_int( "values1" );
  
  // int *a = &values[0];
  // int *b = &values[1];
  // klee_make_symbolic( a, sizeof(int), "values1" );
  // klee_make_symbolic( b, sizeof(int), "values2" );

  int *dvalues;
  hipMalloc((void **)&dvalues, sizeof(int) * NUM);
  hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);
  
  BitonicKernel<<<1, NUM>>>(dvalues);

#ifndef _SYM
  // here blockDim.x should be NUM; we use this hack for convenience
  for (int i = 1; i < NUM; i++) {
    if (dvalues[i] < dvalues[i-1]) {
      printf("The sorting algorithm is incorrect since values[%d] < values[%d]!\n", i, i-1);
      return 1;
    }
  }
#endif

  hipFree(dvalues);
  free(values);

  return 0;
}
