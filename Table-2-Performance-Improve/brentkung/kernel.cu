#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#ifndef N
#error N must be defined
#endif

#if rwidth == 8
  #define rtype unsigned char
  #define MAX_RTYPE 0xff
#elif rwidth == 16
  #define rtype unsigned short
  #define MAX_RTYPE 0xffff
#elif rwidth == 32
  #define rtype unsigned int
  #define MAX_RTYPE 0xffffffff
#elif rwidth == 64
  #define rtype unsigned long
  #define MAX_RTYPE 0xffffffffffffffff
#else
  #error rwidth must be defined
#endif

#ifdef BINOP_ADD
  #define OP(x,y) (x <= MAX_RTYPE - y ? (x + y) : MAX_RTYPE)
#elif BINOP_OR
  #define OP(x,y) (x | y)
#elif BINOP_MAX
  #define OP(x,y) (x < y ? y : x)
#else
  #error Must define one of BINOP_ADD|BINOP_OR|BINOP_MAX
#endif

#ifdef _SYM
  #define ASSERT_NO_CUDA_ERROR( callReturningErrorstatus ) callReturningErrorstatus
#else
  #define ASSERT_NO_CUDA_ERROR( callReturningErrorstatus ) {     \
    hipError_t err = callReturningErrorstatus;                  \
    if (err != hipSuccess) {                                    \
      fprintf(stderr,                                            \
              "Cuda error (%s/%d) in file '%s' in line %i\n",    \
              hipGetErrorString(err), err, __FILE__, __LINE__); \
      exit(1);                                                   \
    }                                                            \
  } while(0);
#endif

__global__ void brentkung(rtype *len, rtype *out) {
  __shared__ rtype result[N];

  unsigned offset;
  unsigned t = threadIdx.x;

  if (t < N/2) {
    result[2*t]   = len[2*t];
    result[2*t+1] = len[2*t+1];
  }

  offset = 1;
  for (unsigned d = N/2; d > 0; d /= 2) {
    __syncthreads();
    if (t < d) {
      unsigned ai = offset * (2 * t + 1) - 1;
      unsigned bi = offset * (2 * t + 2) - 1;
      result[bi] = OP(result[ai],result[bi]);
    }
    offset *= 2;
  }

  for (unsigned d = 2; d < N; d <<= 1) {
    offset >>= 1;
    __syncthreads();
    if (t < (d - 1)) {
      unsigned ai = (offset * (t + 1)) - 1;
      unsigned bi = ai + (offset >> 1);
      result[bi] = OP(result[ai], result[bi]);
    }
  }
  __syncthreads();

  if (t < N/2) {
    out[2*t]   = result[2*t];
    out[2*t+1] = result[2*t+1];
  }
}

int main(int argc, char **argv) {
  // test data
  size_t ArraySize = N * sizeof(rtype);
  rtype *in  = (rtype *)malloc(ArraySize);
  rtype *out = (rtype *)malloc(ArraySize);
#ifdef _SYM
  klee_make_symbolic(in, ArraySize, "in");
#else
  for (unsigned i=0; i<N; ++i) {
    in[i] = 101+i;
  }
#endif

  // create arrays on device
  rtype *d_in;
  rtype *d_out;
  ASSERT_NO_CUDA_ERROR(hipMalloc((void **)&d_in, ArraySize));
  ASSERT_NO_CUDA_ERROR(hipMalloc((void **)&d_out, ArraySize));

  // memcpy into arrays
  ASSERT_NO_CUDA_ERROR(hipMemcpy(d_in, in, ArraySize, hipMemcpyHostToDevice));

  // run the kernel
  ASSERT_NO_CUDA_ERROR(hipDeviceSynchronize());
#ifndef _SYM
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pre-kernel error: %s.\n", hipGetErrorString(err));
    return 1;
  }
#endif
  brentkung<<<1,(N/2)>>>(d_in, d_out);
#ifndef _SYM
  ASSERT_NO_CUDA_ERROR(hipDeviceSynchronize());
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Post-kernel Error: %s\n", hipGetErrorString(err));
    return 1;
  }
#endif

  // memcpy back the result
  ASSERT_NO_CUDA_ERROR(hipMemcpy(out, d_out, ArraySize, hipMemcpyDeviceToHost));

  // check monotonic specification
#ifdef _SYM
  unsigned i,j;
  klee_make_symbolic(&i, sizeof(unsigned), "i");
  klee_make_symbolic(&j, sizeof(unsigned), "j");
  klee_assume(i < N);
  klee_assume(j < N);
  klee_assume(i < j);
  if (!( OP(out[i], in[i+1]) <= out[j] )) {
    printf("TEST FAIL: MONOTONIC SPECIFICATION\n");
    assert(false);
  }
#else
  // check full specification
  rtype sum = in[0];
  for (unsigned i=0; i<N; ++i) {
    printf("out[%d] = %d (%d)\n", i, out[i], sum);
    assert(sum == out[i]);
    if (i < N-1) sum = OP(sum,in[i+1]);
  }
#endif
  printf("TEST PASSED\n");

  // cleanup
  free(in);
  free(out);
  ASSERT_NO_CUDA_ERROR(hipFree(d_in));
  ASSERT_NO_CUDA_ERROR(hipFree(d_out));
  return 0;
}
