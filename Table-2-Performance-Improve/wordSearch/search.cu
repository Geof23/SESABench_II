#include "hip/hip_runtime.h"
/*
 * search_kernel.cu
 * 
 * Search through an unsorted text file for a specific word or phrase. 
 *
 * http://supercomputingblog.com/cuda/search-algorithm-with-cuda/
 */

#include <cstdlib>
#include "stdio.h"

#ifndef NUM1
#error NUM1 must be defined
#endif

#ifndef NUM2
#error NUM2 must be defined
#endif

#ifdef REPLAY

#include "search.cc"

#else

__global__ void wordSearchSimple(char *pszData, int dataLength, 
                                 char *pszTarget, int targetLen, int *pFound) {
  int startIndex = blockDim.x*blockIdx.x + threadIdx.x;
  if (*pFound > startIndex) {
    // only continue if an earlier instance hasn't already been found
    int fMatch = 1;
    for (int i=0; i < targetLen; i++) {
      if (pszData[startIndex+i] != pszTarget[i]) 
        fMatch = 0;
      }
      if (fMatch) 
        atomicMin(pFound, startIndex);
  }
}

#endif

// *************************************************************
//  Driver 
// *************************************************************

int main() {
  char *h_data;
  char *h_target;
  h_data = (char*)malloc(sizeof(char)*NUM1);
  h_target = (char*)malloc(sizeof(char)*NUM2);

  klee_make_symbolic(h_data, sizeof(char) * NUM1, "h_data_input");
  klee_make_symbolic(h_target, sizeof(char) * NUM2, "h_target_input");

  char *d_data;
  char *d_target;
  hipMalloc((void**)&d_data, sizeof(char)*NUM1);
  hipMalloc((void**)&d_target, sizeof(char)*NUM2);
  hipMemcpy(d_data, h_data, sizeof(char) * NUM1, hipMemcpyHostToDevice);
  hipMemcpy(d_target, h_target, sizeof(char) * NUM2, hipMemcpyHostToDevice);

  int found = NUM1;
  
  // the following is equivalent to calling the kernel using <<<...>>>(BitonicKernel)
  wordSearchSimple<<<1, NUM1-NUM2>>>(d_data, sizeof(char)*NUM1, d_target, sizeof(char)*NUM2, &found);

  return 0;
}
