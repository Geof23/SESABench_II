#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include "choose.h"

//#define N 4 
#define MAXWRITE 4

__global__ void compact(int *out, int*in) {
  __shared__ unsigned num[N];
  __shared__ unsigned idx[N];

  unsigned t = threadIdx.x;

  // (i) number of times to repeat element
  num[t] = CHOOSE(in[t], MAXWRITE);

  // (ii) compute indexes for scatter
  //      using an exclusive prefix sum
  __syncthreads();
  if (t < N/2) {
    idx[2*t]   = num[2*t];
    idx[2*t+1] = num[2*t+1];
  }
  // (a) upsweep
  int offset = 1;
  for (unsigned d = N/2; d > 0; d /= 2) {
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      idx[bi] += idx[ai];
    }
    offset *= 2;
  }
  // (b) downsweep
  if (t == 0) idx[N-1] = 0;
  for (unsigned d = 1; d < N; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      int temp = idx[ai];
      idx[ai] = idx[bi];
      idx[bi] += temp;
    }
  }
  __syncthreads();
  // end of exclusive prefix sum of flag into idx

  // (iii) repeat element num times
  for (unsigned i = 0; i < num[t]; ++i) {
    out[idx[t]+i] = in[t];
  }
}

int main(int argc, char **argv) {
  // test data
  size_t ArraySize = N * sizeof(int);
  size_t OutArraySize = (MAXWRITE-1) * N * sizeof(int);
  int *in  = (int *)malloc(ArraySize);
  int *out = (int *)malloc(OutArraySize);
  klee_make_symbolic(in, ArraySize, "in");

  // create some memory objects on the device
  int *d_in;
  int *d_out;
  hipMalloc((void **)&d_in, ArraySize);
  hipMalloc((void **)&d_out, ArraySize);

  // memcpy into these objects
  hipMemcpy(d_in, in, ArraySize, hipMemcpyHostToDevice);

  // run the kernel
  compact<<<1,N>>>(d_out, d_in);

  // memcpy back the result
  hipMemcpy(out, d_out, OutArraySize, hipMemcpyDeviceToHost);

#ifndef _SYM
  // check results
  unsigned idx = 0;
  for (unsigned i=0; i<N; ++i) {
    unsigned num = CHOOSE(in[i], MAXWRITE);
    for (unsigned j=0; j<num; ++j) {
      assert(out[idx+j] == in[i]);
    }
    idx += num;
  }
  printf("TEST PASSED\n");
#endif

  // cleanup
  free(in);
  free(out);
  hipFree(d_in);
  hipFree(d_out);
  return 0;
}
