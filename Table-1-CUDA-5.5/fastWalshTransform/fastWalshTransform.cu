#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Walsh transforms belong to a class of generalized Fourier transformations.
 * They have applications in various fields of electrical engineering
 * and numeric theory. In this sample we demonstrate efficient implementation
 * of naturally-ordered Walsh transform
 * (also known as Walsh-Hadamard or Hadamard transform) in CUDA and its
 * particular application to dyadic convolution computation.
 * Refer to excellent Jorg Arndt's "Algorithms for Programmers" textbook
 * http://www.jjj.de/fxt/fxtbook.pdf (Chapter 22)
 *
 * Victor Podlozhnyuk (vpodlozhnyuk@nvidia.com)
 */



#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>


////////////////////////////////////////////////////////////////////////////////
// Reference CPU FWT
////////////////////////////////////////////////////////////////////////////////
extern"C" void fwtCPU(float *h_Output, float *h_Input, int log2N);
extern"C" void slowWTcpu(float *h_Output, float *h_Input, int log2N);
extern "C" void dyadicConvolutionCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int log2dataN,
    int log2kernelN
);


////////////////////////////////////////////////////////////////////////////////
// GPU FWT
////////////////////////////////////////////////////////////////////////////////
//#include "fastWalshTransform_kernel.cuh"

#define ELEMENTARY_LOG2SIZE 11

__extern__shared__ float s_data[];
__global__ void fwtBatch1Kernel(float *d_Output, float *d_Input, int log2N)
{
    const int    N = 1 << log2N;
    const int base = blockIdx.x << log2N;

    //(2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
    float *d_Src = d_Input  + base;
    float *d_Dst = d_Output + base;

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        s_data[pos] = d_Src[pos];
    }

    //Main radix-4 stages
    const int pos = threadIdx.x;

    for (int stride = N >> 2; stride > 0; stride >>= 2)
    {
        int lo = pos & (stride - 1);
        int i0 = ((pos - lo) << 2) + lo;
        int i1 = i0 + stride;
        int i2 = i1 + stride;
        int i3 = i2 + stride;

        __syncthreads();
        float D0 = s_data[i0];
        float D1 = s_data[i1];
        float D2 = s_data[i2];
        float D3 = s_data[i3];

        float T;
        T = D0;
        D0         = D0 + D2;
        D2         = T - D2;
        T = D1;
        D1         = D1 + D3;
        D3         = T - D3;
        T = D0;
        s_data[i0] = D0 + D1;
        s_data[i1] = T - D1;
        T = D2;
        s_data[i2] = D2 + D3;
        s_data[i3] = T - D3;
    }

    //Do single radix-2 stage for odd power of two
    if (log2N & 1)
    {
        __syncthreads();

        for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x)
        {
            int i0 = pos << 1;
            int i1 = i0 + 1;

            float D0 = s_data[i0];
            float D1 = s_data[i1];
            s_data[i0] = D0 + D1;
            s_data[i1] = D0 - D1;
        }
    }

    __syncthreads();

    for (int pos = threadIdx.x; pos < N; pos += blockDim.x)
    {
        d_Dst[pos] = s_data[pos];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
__global__ void fwtBatch2Kernel(
    float *d_Output,
    float *d_Input,
    int stride
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int   N = blockDim.x *  gridDim.x * 4;

    float *d_Src = d_Input  + blockIdx.y * N;
    float *d_Dst = d_Output + blockIdx.y * N;

    int lo = pos & (stride - 1);
    int i0 = ((pos - lo) << 2) + lo;
    int i1 = i0 + stride;
    int i2 = i1 + stride;
    int i3 = i2 + stride;

    float D0 = d_Src[i0];
    float D1 = d_Src[i1];
    float D2 = d_Src[i2];
    float D3 = d_Src[i3];

    float T;
    T = D0;
    D0        = D0 + D2;
    D2        = T - D2;
    T = D1;
    D1        = D1 + D3;
    D3        = T - D3;
    T = D0;
    d_Dst[i0] = D0 + D1;
    d_Dst[i1] = T - D1;
    T = D2;
    d_Dst[i2] = D2 + D3;
    d_Dst[i3] = T - D3;
}

////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(float *d_Data, int M, int log2N)
{
    const int THREAD_N = 256;

    int N = 1 << log2N;
    dim3 grid((1 << log2N) / (4 * THREAD_N), M, 1);

    for (; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, N >>= 2, M <<= 2)
    {
        fwtBatch2Kernel<<<grid, THREAD_N>>>(d_Data, d_Data, N / 4);
        //getLastCudaError("fwtBatch2Kernel() execution failed\n");
        printf("fwtBatch2Kernel() execution failed\n");
    }

    fwtBatch1Kernel<<<M, N / 4, N *sizeof(float)>>>(
        d_Data,
        d_Data,
        log2N
    );
    //getLastCudaError("fwtBatch1Kernel() execution failed\n");
    printf("fwtBatch1Kernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
__global__ void modulateKernel(float *d_A, float *d_B, int N)
{
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    float     rcpN = 1.0f / (float)N;

    for (int pos = tid; pos < N; pos += numThreads)
    {
        d_A[pos] *= d_B[pos] * rcpN;
    }
}

//Interface to modulateKernel()
void modulateGPU(float *d_A, float *d_B, int N)
{
    modulateKernel<<<128, 256>>>(d_A, d_B, N);
}


////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int log2Kernel = 7;
//const   int log2Data = 23;
const int log2Kernel = 6;
const   int log2Data = 12;

const int   dataN = 1 << log2Data;
const int kernelN = 1 << log2Kernel;

const int   DATA_SIZE = dataN   * sizeof(float);
const int KERNEL_SIZE = kernelN * sizeof(float);

const long NOPS = 3.0 * (double)dataN * (double)log2Data / 2.0;
//const double NOPS = 3.0 * (double)dataN * (double)log2Data / 2.0;



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{
    float *h_Data,
          *h_Kernel,
          *h_ResultCPU,
          *h_ResultGPU;

    float *d_Data,
          *d_Kernel;

    double delta, ref, sum_delta2, sum_ref2, L2norm, gpuTime;

    //StopWatchInterface *hTimer = NULL;
    int i;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);

    //sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory\n");
    h_Kernel    = (float *)malloc(KERNEL_SIZE);
    h_Data      = (float *)malloc(DATA_SIZE);
    h_ResultCPU = (float *)malloc(DATA_SIZE);
    h_ResultGPU = (float *)malloc(DATA_SIZE);
    printf("...allocating GPU memory\n");
    //checkCudaErrors(hipMalloc((void **)&d_Kernel, DATA_SIZE));
    //checkCudaErrors(hipMalloc((void **)&d_Data,   DATA_SIZE));
    hipMalloc((void **)&d_Kernel, DATA_SIZE);
    hipMalloc((void **)&d_Data,   DATA_SIZE);

    printf("...generating data\n");
    printf("Data length: %i; kernel length: %i\n", dataN, kernelN);
#ifdef _SYM
    klee_make_symbolic(h_Kernel, KERNEL_SIZE, "h_Kernel_input");
    klee_make_symbolic(h_Data, DATA_SIZE, "h_Data_input");
#else
    srand(2007);

    for (i = 0; i < kernelN; i++)
    {
        h_Kernel[i] = (float)rand() / (float)RAND_MAX;
    }

    for (i = 0; i < dataN; i++)
    {
        h_Data[i] = (float)rand() / (float)RAND_MAX;
    }
#endif

    //checkCudaErrors(hipMemset(d_Kernel, 0, DATA_SIZE));
    //checkCudaErrors(hipMemcpy(d_Kernel, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(d_Data,   h_Data,     DATA_SIZE, hipMemcpyHostToDevice));
    hipMemset(d_Kernel, 0, DATA_SIZE);
    hipMemcpy(d_Kernel, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_Data,   h_Data,     DATA_SIZE, hipMemcpyHostToDevice);

    printf("Running GPU dyadic convolution using Fast Walsh Transform...\n");
    //checkCudaErrors(hipDeviceSynchronize());
    //sdkResetTimer(&hTimer);
    //sdkStartTimer(&hTimer);
    fwtBatchGPU(d_Data, 1, log2Data);
    fwtBatchGPU(d_Kernel, 1, log2Data);
    modulateGPU(d_Data, d_Kernel, dataN);
    fwtBatchGPU(d_Data, 1, log2Data);
    //checkCudaErrors(hipDeviceSynchronize());
    //sdkStopTimer(&hTimer);
    //gpuTime = sdkGetTimerValue(&hTimer);
    // removed for global FP choking Gklee
    //    printf("GPU time: %f ms; GOP/s: %f\n", gpuTime, NOPS / (gpuTime * 0.001 * 1E+9));

    printf("Reading back GPU results...\n");
    //checkCudaErrors(hipMemcpy(h_ResultGPU, d_Data, DATA_SIZE, hipMemcpyDeviceToHost));
    hipMemcpy(h_ResultGPU, d_Data, DATA_SIZE, hipMemcpyDeviceToHost);

    printf("Running straightforward CPU dyadic convolution...\n");
    //dyadicConvolutionCPU(h_ResultCPU, h_Data, h_Kernel, log2Data, log2Kernel);

    printf("Comparing the results...\n");
    sum_delta2 = 0;
    sum_ref2   = 0;

    /*for (i = 0; i < dataN; i++)
    {
        delta       = h_ResultCPU[i] - h_ResultGPU[i];
        ref         = h_ResultCPU[i];
        sum_delta2 += delta * delta;
        sum_ref2   += ref * ref;
    }*/

    L2norm = sqrt(sum_delta2 / sum_ref2);

    printf("Shutting down...\n");
    //sdkDeleteTimer(&hTimer);
    //checkCudaErrors(hipFree(d_Data));
    //checkCudaErrors(hipFree(d_Kernel));
    hipFree(d_Data);
    hipFree(d_Kernel);
    free(h_ResultGPU);
    free(h_ResultCPU);
    free(h_Data);
    free(h_Kernel);

    //hipDeviceReset();
    printf("L2 norm: %E\n", L2norm);
    printf(L2norm < 1e-6 ? "Test passed\n" : "Test failed!\n");
}
